#include "parameter_classes.h"
#include "utils.h"

VolumeCfg::VolumeCfg(int d, int h, int w, float _dz, float _dy, float _dx, bool ddd)
        : depth(d), height(h), width(w), dz(_dz), dy(_dy), dx(_dx), is_3d(ddd) {}

ProjectionCfg::ProjectionCfg(int dc_u, float ds_u, int dc_v, float ds_v, float sd, float dd,
                  float pi, float iz, int pt)
            : det_count_u(dc_u), det_spacing_u(ds_u), det_count_v(dc_v),
              det_spacing_v(ds_v), s_dist(sd), d_dist(dd), pitch(pi), initial_z(iz),
              projection_type(pt) {}

ProjectionCfg::ProjectionCfg(const ProjectionCfg& src)
    :det_count_u(src.det_count_u), det_spacing_u(src.det_spacing_u),
    det_count_v(src.det_count_v), det_spacing_v(src.det_spacing_v),
    s_dist(src.s_dist), d_dist(src.d_dist), pitch(src.pitch), initial_z(src.initial_z),
              projection_type(src.projection_type), n_angles(src.n_angles) {}

bool ProjectionCfg::is_2d() const{
    return projection_type == PARALLEL || projection_type == FANBEAM;
}

ProjectionCfg ProjectionCfg::copy() const{
    return ProjectionCfg(*this);
}

//std::string ProjectionCfg::to_string() const{
//    switch(projection_type){
//        case PARALLEL:
//            return string_format("Projection(type=ParallelBeam, det_count=%d, det_spacing=%f)", det_count_u, det_spacing_u);
//        default:
//            return "TODO";
//    }
//}


ExecCfg::ExecCfg(int x, int y, int z, int ch)
        :block_dim(x, y, z), channels(ch) {}

dim3 ExecCfg::get_grid_size(int x, int y, int z) const{
    return dim3(roundup_div(x, block_dim.x), roundup_div(y, block_dim.y), roundup_div(z, block_dim.z));
}

int ExecCfg::get_channels(int batch_size) const{
    return (batch_size % 4 == 0) ? this->channels : 1;
}