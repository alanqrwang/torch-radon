#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>


#include "utils.h"
#include "texture.h"

template<bool parallel_beam, int channels, bool clip_to_circle, typename T>
__global__ void
radon_forward_kernel(T *__restrict__ output, hipTextureObject_t texture, const float *__restrict__ angles,
                     RaysCfg cfg) {
    // Calculate texture coordinates
    const int ray_id = blockIdx.x * blockDim.x + threadIdx.x;
    const int angle_id = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_id = blockIdx.z * channels;

    if (angle_id < cfg.n_angles && ray_id < cfg.det_count) {
        
        float accumulator[channels];

        #pragma unroll
        for (int i = 0; i < channels; i++) {
            accumulator[i] = 0.0f;
        }


        // compute ray
        float v, sx, sy, ex, ey;
        if (parallel_beam) {
            v = cfg.height / 2.0;
            sx = (ray_id - cfg.det_count / 2.0f + 0.5f) * cfg.det_spacing;
            sy = 0.71f * cfg.height;
            ex = sx;
            ey = -sy;
        } else {
            v = cfg.height / 2.0;
            sy = cfg.s_dist;
            sx = 0.0f;
            ey = -cfg.d_dist;
            ex = (ray_id - cfg.det_count / 2.0f + 0.5f) * cfg.det_spacing;
        }

        
        // rotate ray
        const float angle = angles[angle_id];
        const float cs = __cosf(angle);
        const float sn = __sinf(angle);

        // start position rs and direction rd
        float rsx = sx * cs - sy * sn;
        float rsy = sx * sn + sy * cs;
        float rdx = ex * cs - ey * sn - rsx;
        float rdy = ex * sn + ey * cs - rsy;


        if (cfg.clip_to_circle) {
            // clip rays to circle (to reduce memory reads)
            const float radius = cfg.det_count / 2.0f;
            const float a = rdx * rdx + rdy * rdy;
            const float b = rsx * rdx + rsy * rdy;
            const float c = rsx * rsx + rsy * rsy - radius * radius;

            // min_clip to 1 to avoid getting empty rays
            const float delta_sqrt = sqrtf(max(b * b - a * c, 1.0f));
            const float alpha_s = (-b - delta_sqrt) / a;
            const float alpha_e = (-b + delta_sqrt) / a;

            rsx += rdx * alpha_s + v;
            rsy += rdy * alpha_s + v;
            rdx *= (alpha_e - alpha_s);
            rdy *= (alpha_e - alpha_s);
        } else {
            // clip to square (to reduce memory reads)
            const float alpha_x_m = (-v - rsx)/rdx;
            const float alpha_x_p = (v - rsx)/rdx;
            const float alpha_y_m = (-v -rsy)/rdy;
            const float alpha_y_p = (v - rsy)/rdy;
            const float alpha_s = max(min(alpha_x_p, alpha_x_m), min(alpha_y_p, alpha_y_m));
            const float alpha_e = min(max(alpha_x_p, alpha_x_m), max(alpha_y_p, alpha_y_m));

            if(alpha_s > alpha_e){
                #pragma unroll
                for (int b = 0; b < channels; b++) {
                    output[(batch_id + b) * cfg.det_count * cfg.n_angles + angle_id * cfg.det_count + ray_id] = 0.0f;
                }
                return;
            }

            rsx += rdx*alpha_s + v;
            rsy += rdy*alpha_s + v;
            rdx *= (alpha_e - alpha_s);
            rdy *= (alpha_e - alpha_s);
        }
        
        const uint n_steps = __float2uint_ru(hypot(rdx, rdy));
        const float vx = rdx / n_steps;
        const float vy = rdy / n_steps;
        const float n = hypot(vx, vy);

        for (uint j = 0; j <= n_steps; j++) { //changing j and n_steps to int makes everything way slower (WHY???)
            if (channels == 1) {
                accumulator[0] += tex2DLayered<float>(texture, rsx, rsy, blockIdx.z);
            } else {
                float4 read = tex2DLayered<float4>(texture, rsx, rsy, blockIdx.z);
                accumulator[0] += read.x;
                accumulator[1] += read.y;
                accumulator[2] += read.z;
                accumulator[3] += read.w;
            }
            rsx += vx;
            rsy += vy;
        }


        
        #pragma unroll
        for (int b = 0; b < channels; b++) {
            output[(batch_id + b) * cfg.det_count * cfg.n_angles + angle_id * cfg.det_count + ray_id] =
                    accumulator[b] * n;
        }

    }
}

template<typename T>
void radon_forward_cuda(
        const T *x, const float *angles, T *y,
        TextureCache &tex_cache, const RaysCfg &cfg, const int batch_size, const int device
) {
//    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(radon_forward_kernel_half<true>), hipFuncCachePreferL1));
//    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(radon_forward_kernel_half<false>), hipFuncCachePreferL1));

    constexpr bool is_float = std::is_same<T, float>::value;
    constexpr int precision = is_float ? PRECISION_FLOAT : PRECISION_HALF;
    const int channels = (batch_size % 4 == 0) ? 4 : 1;

    // copy x into CUDA Array (allocating it if needed) and bind to texture
    Texture *tex = tex_cache.get({device, batch_size, cfg.height, cfg.width, 0, channels, precision});
    tex->put(x);

    // Invoke kernel
    dim3 block_dim(16, 16);
    dim3 grid_dim(roundup_div(cfg.det_count, 16), roundup_div(cfg.n_angles, 16), batch_size / channels);

    if (cfg.is_fanbeam()) {
        if (channels == 1) {
            if (cfg.clip_to_circle) {
                radon_forward_kernel<false, 1, true> << < grid_dim, block_dim >> > (y, tex->texture, angles, cfg);
            } else {
                radon_forward_kernel<false, 1, false> << < grid_dim, block_dim >> > (y, tex->texture, angles, cfg);
            }
        } else {
            if (is_float) {
                if (cfg.clip_to_circle) {
                    radon_forward_kernel<false, 4, true> << < grid_dim, block_dim >> > (y, tex->texture, angles, cfg);
                } else {
                    radon_forward_kernel<false, 4, false> << < grid_dim, block_dim >> > (y, tex->texture, angles, cfg);
                }
            } else {
                if (cfg.clip_to_circle) {
                    radon_forward_kernel<false, 4, true> << < grid_dim, block_dim >> >
                                                                        ((__half *) y, tex->texture, angles, cfg);
                } else {
                    radon_forward_kernel<false, 4, false> << < grid_dim, block_dim >> >
                                                                         ((__half *) y, tex->texture, angles, cfg);
                }
            }
        }
    } else {
        if (channels == 1) {
            if (cfg.clip_to_circle) {
                radon_forward_kernel<true, 1, true> << < grid_dim, block_dim >> > (y, tex->texture, angles, cfg);
            } else {
                radon_forward_kernel<true, 1, false> << < grid_dim, block_dim >> > (y, tex->texture, angles, cfg);
            }
        } else {
            if (is_float) {
                if (cfg.clip_to_circle) {
                    radon_forward_kernel<true, 4, true> << < grid_dim, block_dim >> > (y, tex->texture, angles, cfg);
                } else {
                    radon_forward_kernel<true, 4, false> << < grid_dim, block_dim >> > (y, tex->texture, angles, cfg);
                }
            } else {
                if (cfg.clip_to_circle) {
                    radon_forward_kernel<true, 4, true> << < grid_dim, block_dim >> >
                                                                       ((__half *) y, tex->texture, angles, cfg);
                } else {
                    radon_forward_kernel<true, 4, false> << < grid_dim, block_dim >> >
                                                                        ((__half *) y, tex->texture, angles, cfg);
                }
            }
        }
    }
}

template void
radon_forward_cuda<float>(const float *x, const float *angles, float *y, TextureCache &tex_cache, const RaysCfg &cfg,
                          const int batch_size, const int device);

template void radon_forward_cuda<unsigned short>(const unsigned short *x, const float *angles, unsigned short *y,
                                                 TextureCache &tex_cache, const RaysCfg &cfg,
                                                 const int batch_size, const int device);


template<int channels, typename T>
__global__ void
radon_forward_kernel_3d(T *__restrict__ output, hipTextureObject_t texture, const float *__restrict__ angles,
                     RaysCfg cfg) {
    // Calculate sensor coordinates in pixels
    const int pu = blockIdx.x * blockDim.x + threadIdx.x;
    const int angle_id = blockIdx.y * blockDim.y + threadIdx.y;
    const int pv = blockIdx.z * blockDim.z + threadIdx.z;


    if (angle_id < cfg.n_angles && pu < cfg.det_count && pv < cfg.det_count_z) {
        float accumulator = 0.0f;

        // compute ray
        const float angle = angles[angle_id];
        const float cs = __cosf(angle);
        const float sn = __sinf(angle);

        float sx = 0.0f;
        float sy = -cfg.s_dist;
        // sz = initial_z + pitch * angle / (2*pi);
        float rsz = cfg.initial_z + cfg.pitch * angle * 0.1591549f;

        float ex = (pu - cfg.det_count * 0.5f + 0.5f) * cfg.det_spacing;
        float ey = cfg.d_dist;
        float rdz = (pv - cfg.det_count_z * 0.5f + 0.5f) * cfg.det_spacing_z;

        // rotate start position rs and direction rd
        float rsx = sx * cs - sy * sn;
        float rsy = sx * sn + sy * cs;
        float rdx = ex * cs - ey * sn - rsx;
        float rdy = ex * sn + ey * cs - rsy;

        rsx += cfg.width * 0.5f;
        rsy += cfg.height * 0.5f;

        // Clip ray to cube (with a little bit of padding) to reduce the number of memory reads
        constexpr float pad = 1.0f;
        const float alpha_x_m = (-pad - rsx)/rdx;
        const float alpha_x_p = (cfg.width + pad - rsx)/rdx;
        const float alpha_y_m = (-pad - rsy)/rdy;
        const float alpha_y_p = (cfg.height + pad - rsy)/rdy;
        const float alpha_z_m = (-pad - rsz)/rdz;
        const float alpha_z_p = (cfg.depth + pad - rsz)/rdz;

        const float alpha_s = max(min(alpha_x_p, alpha_x_m), max(min(alpha_y_p, alpha_y_m), min(alpha_z_p, alpha_z_m)));
        const float alpha_e = min(max(alpha_x_p, alpha_x_m), min(max(alpha_y_p, alpha_y_m), max(alpha_z_p, alpha_z_m)));

        if(alpha_s > alpha_e){
            output[angle_id * cfg.det_count_z * cfg.det_count + pv * cfg.det_count + pu] = 0.0f;
            return;
        }

        rsx += rdx*alpha_s;
        rsy += rdy*alpha_s;
        rsz += rdz*alpha_s;
        rdx *= (alpha_e - alpha_s);
        rdy *= (alpha_e - alpha_s);
        rdz *= (alpha_e - alpha_s);

        // accumulate loop
        const uint n_steps = __float2uint_ru( norm3df(rdx, rdy, rdz));
        const float vx = rdx / n_steps;
        const float vy = rdy / n_steps;
        const float vz = rdz / n_steps;
        const float n = norm3df(vx, vy, vz);

        for (uint j = 0; j <= n_steps; j++) { //changing j and n_steps to int makes everything way slower (WHY???)
            accumulator += tex3D<float>(texture, rsx, rsy, rsz);

            rsx += vx;
            rsy += vy;
            rsz += vz;
        }

        // output
        output[angle_id * cfg.det_count_z * cfg.det_count + pv * cfg.det_count + pu] = accumulator * n;
    }
}

void radon_forward_cuda_3d(
        const float *x, const float *angles, float *y,
        TextureCache &tex_cache, const RaysCfg &cfg, const int batch_size, const int device
){
    constexpr bool is_float = true;
    constexpr int precision = is_float ? PRECISION_FLOAT : PRECISION_HALF;
    const int channels = 1; //(batch_size % 4 == 0) ? 4 : 1;

    // copy x into CUDA Array (allocating it if needed) and bind to texture
    Texture *tex = tex_cache.get({device, 0, cfg.height, cfg.width, cfg.depth, channels, precision});
    tex->put(x);

    // Invoke kernel
    dim3 block_dim(8, 8, 8);
    dim3 grid_dim(
        roundup_div(cfg.det_count, block_dim.x),
        roundup_div(cfg.n_angles, block_dim.y),
        roundup_div(cfg.det_count_z, block_dim.z)
    );

    radon_forward_kernel_3d<1> <<< grid_dim, block_dim >>> (y, tex->texture, angles, cfg);
}